#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "timing.h"

using namespace std;

#define from2Dto1D(x, y, length) ((y)*length+(x))
#define WARP_SIZE 32
#define FULL_MASK 0xffffffff

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/* CPU Functions */
void stencil(int **dev_input, int **dev_output, int size, int stride, int length, int time, float selfCoefficient, float neighborCoefficient);
__global__ void run_single_stencil(int *dev_input, int *dev_output, const int C, int offset_tile_x, int offset_tile_y, int length, int stride, int P, float selfCoefficient, float neighborCoefficient);
void read_input(int **input, int **output, string filename, int length);
void write_output(int *output, string filename, int length);

/* GPU Functions */
void copy_input_to_gpu(int *input, int **dev_input, int **dev_output, int length);

int main(int argc, char *argv[])
{
  /* Define variables */
  int *input, *output;
  int *dev_input, *dev_output;
  int size, stride, length, time;
  string filename, output_filename;
  MyTimer timer;
  float selfCoefficient = 1.0/9.0;
  float neighborCoefficient = 1.0/9.0;

  /* Check if the arguments are set */
  if(argc < 4) {
    cerr << "Usage: ./CPU <size> <stride> <time>\n";
    exit(EXIT_FAILURE);
  }

  /* Set initial variables */
  size             = atoi(argv[1]);
  stride           = atoi(argv[2]);
  time             = atoi(argv[3]);
  length           = 1 << size;       // length = 2 ^ size
  length          += 2 * stride;      //        + 2 * stride
  filename         = "../../Data/data_";
  filename        += to_string(size) + "_" + to_string(stride) + ".dat";
  output_filename  = "../../Data/gpu_shfl_";
  output_filename += to_string(size) + "_" + to_string(stride) + ".dat";

  /* Read data from input file */
  read_input(&input, &output, filename, length);

  /* Copy data to GPU */
  copy_input_to_gpu(input, &dev_input, &dev_output, length);

  /* Run Stencil */
  timer.StartTimer();
  stencil(&dev_input, &dev_output, size, stride, length, time, selfCoefficient, neighborCoefficient);
  hipDeviceSynchronize();
  timer.StopTimer();

  /* Print duration */
  cout << "It took " << timer.GetDurationInSecondsAccurate() << " seconds to run!\n";

  /* Copy data back to CPU */
  gpuErrchk(hipMemcpy(output, dev_output, length * length * sizeof(int), hipMemcpyDeviceToHost));

  /* Output data */
  write_output(output, output_filename, length);

  /* Free allocated memory */
  hipFree(dev_input);
  hipFree(dev_output);
  delete [] input;
  delete [] output;

  /* End of program */
  return 0;
}

void stencil(int **dev_input, int **dev_output, int size, int stride, int length, int time, float selfCoefficient, float neighborCoefficient)
{
  /* Define variables */
  int i;
  int **swap;

  /* System variables */
  int P                 = 10;                // P: defines the number of cell each thread calculates
  int N                 = (2 * stride) + 1;  // N: stencil length each direction
  int C                 = (N + P - 1);       // C: each block will calculate (warp_size * C) size

  int number_of_tiles_x = int(length / (WARP_SIZE - 2 * stride)) + 1;
  int number_of_tiles_y = int(length / P) + 1;
  int offset_tile_x     = WARP_SIZE - 2 * stride;
  int offset_tile_y     = P;

  /* Loop over time dimension */
  for(i=0; i<time; i++) {
    /* Calculate block and grid sizes */
    dim3 block_size = dim3(WARP_SIZE, 1, 1);
    dim3 grid_size = dim3(number_of_tiles_x, number_of_tiles_y, 1);
    run_single_stencil<<< grid_size, block_size >>>(*dev_input, *dev_output, C, offset_tile_x, offset_tile_y, length, stride, P, selfCoefficient, neighborCoefficient);
    gpuErrchk(hipGetLastError());
    hipDeviceSynchronize();

    /* Swap pointers after each run so dev_output will always be output,
     * and dev_input will be always input
     */
    swap = dev_input;
    dev_input = dev_output;
    dev_output = swap;
  }
}

__global__ void run_single_stencil(int *dev_input, int *dev_output, const int C, int offset_tile_x, int offset_tile_y, int length, int stride, int P, float selfCoefficient, float neighborCoefficient)
{
  /* Declare variables */
  int i, j;
  int v[14], o[14]; // C=N+P-1
  int offset_x = blockIdx.x * offset_tile_x;
  int offset_y = blockIdx.y * offset_tile_y;
  int lane     = threadIdx.x;

  /* Initialize v[] array */
  for(i=0; i<C; i++) {
    v[i] = dev_input[from2Dto1D(lane + offset_x, i + offset_y, length)];
  }

  /* Main loop calculates for all P elements */
  for(i=stride; i<P+stride; i++) {
    int sum = 0;

    /* Left wing */
    for(j=-stride; j<0; j++) {
      //sum = v[i] * neighborCoefficient + sum;
      sum = v[i] + sum;

      /* Shuffle up */
      sum = __shfl_up_sync(FULL_MASK, sum, 1);
    }

    /* Center column */
    for(j=-stride; j<=stride; j++) {
      sum = v[i] + sum;
    }

    /* Right wing */
    for(j=1; j<=stride; j++) {
      /* Shuffle up */
      sum = __shfl_up_sync(FULL_MASK, sum, 1);
      
      sum = v[i] + sum;
    }
    
    sum /= (stride * 4 + 1);
    o[i] = sum;
  }

  /* Write the sum back to global memory */
  for(i=stride; i<P+stride; i++) {
    dev_output[from2Dto1D(lane + offset_x - stride, i + offset_y, length)] = o[i];
    if(threadIdx.x == 0 && blockIdx.x == 0)
      printf("x: %d, y: %d\n", lane+offset_x-stride, i+offset_y);
  }
}

void read_input(int **input, int **output, string filename, int length)
{
  /* Define variables */
  int i, j;
  ifstream fp;

  /* Open input file */
  fp.open(filename.c_str(), ios::in);
  if(!fp) {
    cerr << "Couldn't open input file to read data!\n";
    exit(EXIT_FAILURE);
  }

  /* Allocate space for our arrays */
  *input = new int[length * length];
  *output = new int[length * length];

  /* Read data from file */
  for(i=0; i<length; i++) {
    for(j=0; j<length; j++) {
      fp >> (*input)[from2Dto1D(i, j, length)];
    }
  }
}

void write_output(int *output, string filename, int length)
{
  /* Define variables */
  int i, j;
  ofstream fp;

  /* Open output file */
  fp.open(filename, ios::out);
  if(!fp) {
    cerr << "Couldn't open output file to write data!\n";
    exit(EXIT_FAILURE);
  }

  for(i=0; i<length; i++) {
    for(j=0; j<length; j++) {
      fp << output[from2Dto1D(i, j, length)] << " ";
    }
    fp << "\n";
  }
}

void copy_input_to_gpu(int *input, int **dev_input, int **dev_output, int length)
{
  /* Allocate GPU memory for input and output arrays */
  gpuErrchk(hipMalloc((void**) dev_input, length * length * sizeof(int)));
  gpuErrchk(hipMalloc((void**) dev_output, length * length * sizeof(int)));

  /* Copy input array to GPU */
  gpuErrchk(hipMemcpy(*dev_input, input, length * length * sizeof(int), hipMemcpyHostToDevice));
}