#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "timing.h"
#include <algorithm>

using namespace std;

#define from2Dto1D(x, y, length) ((y)*length+(x))
#define WARP_SIZE 32
#define FULL_MASK 0xffffffff
#define DATA_TYPE float
#define NUMBER_OF_WARPS_PER_X 1
#define P 2
#define STRIDE 4
#define N 9        // N = 2 * STRIDE + 1
#define C 10       // C = (N+P-1)

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

string toString(int n)
{
    string tmp = "";
    while (n > 0) {
        int re = n % 10;
        n = n / 10;
        tmp += re + '0';
    }
    reverse(tmp.begin(), tmp.end());
    return tmp;
}

/* CPU Functions */
void stencil(DATA_TYPE **dev_input, DATA_TYPE **dev_output, int size,
  int length, int time, float selfCoefficient, float neighborCoefficient);
__global__ void run_single_stencil(DATA_TYPE *dev_input, DATA_TYPE *dev_output,
  int offset_tile_x, int length, float selfCoefficient,
  float neighborCoefficient);
void read_input(DATA_TYPE **input, DATA_TYPE **output, string filename,
  int length);
void write_output(DATA_TYPE *output, string filename, int length);

/* GPU Functions */
void copy_input_to_gpu(DATA_TYPE *input, DATA_TYPE **dev_input, DATA_TYPE **dev_output, int length);

int main(int argc, char *argv[])
{
  /* Define variables */
  DATA_TYPE *input, *output;
  DATA_TYPE *dev_input, *dev_output;
  int size, length, time;
  string filename, output_filename;
  MyTimer timer;
  float selfCoefficient = 1.0/9.0;
  float neighborCoefficient = 1.0/9.0;

  /* Check if the arguments are set */
  if(argc < 3) {
    cerr << "Usage: ./CPU <size> <time>\n";
    exit(EXIT_FAILURE);
  }

  /* Set initial variables */
  size             = atoi(argv[1]);
  time             = atoi(argv[2]);
  length           = 1 << size;       // length = 2 ^ size
  length          += 2 * STRIDE;      //        + 2 * stride
  filename         = "../../Data/data_";
  filename        += toString(size) + "_" + toString(STRIDE) + ".dat";
  output_filename  = "../../Data/gpu_shfl_";
  output_filename += toString(size) + "_" + toString(STRIDE) + ".dat";

  /* Read data from input file */
  read_input(&input, &output, filename, length);

  /* Copy data to GPU */
  copy_input_to_gpu(input, &dev_input, &dev_output, length);

  /* Run Stencil */
  timer.StartTimer();
  stencil(&dev_input, &dev_output, size, length, time, selfCoefficient, neighborCoefficient);
  hipDeviceSynchronize();
  timer.StopTimer();

  /* Print duration */
  cout << "It took " << timer.GetDurationInSecondsAccurate() << " seconds to run!\n";

  /* Copy data back to CPU */
  gpuErrchk(hipMemcpy(output, dev_output, length * length * sizeof(DATA_TYPE), hipMemcpyDeviceToHost));

  /* Output data */
  write_output(output, output_filename, length);

  /* Free allocated memory */
  hipFree(dev_input);
  hipFree(dev_output);
  delete [] input;
  delete [] output;

  /* End of program */
  return 0;
}

void stencil(DATA_TYPE **dev_input, DATA_TYPE **dev_output, int size,
  int length, int time, float selfCoefficient, float neighborCoefficient)
{
  /* Define variables */
  int i;
  DATA_TYPE **swap;

  int number_of_tiles_x = int(length / (WARP_SIZE * NUMBER_OF_WARPS_PER_X - 2 * STRIDE)) + 1;
  int number_of_tiles_y = int(length / P) + 1;
  int offset_tile_x     = WARP_SIZE - 2 * STRIDE;

  /* Loop over time dimension */
  for(i=0; i<time; i++) {
    /* Calculate block and grid sizes */
    dim3 block_size = dim3(WARP_SIZE * NUMBER_OF_WARPS_PER_X, 1, 1);
    dim3 grid_size = dim3(number_of_tiles_x, number_of_tiles_y, 1);
    run_single_stencil<<< grid_size, block_size >>>(*dev_input, *dev_output,
      offset_tile_x, length, selfCoefficient, neighborCoefficient);
    gpuErrchk(hipGetLastError());
    //hipDeviceSynchronize();

    /* Swap pointers after each run so dev_output will always be output,
     * and dev_input will be always input
     */
    swap = dev_input;
    dev_input = dev_output;
    dev_output = swap;
  }
}

__global__ void run_single_stencil(DATA_TYPE *dev_input, DATA_TYPE *dev_output,
  int offset_tile_x, int length, DATA_TYPE selfCoefficient,
  DATA_TYPE neighborCoefficient)
{
  /* Declare variables */
  int i, j;
  DATA_TYPE v[C], o[C];
  int offset_x = blockIdx.x * (offset_tile_x * NUMBER_OF_WARPS_PER_X) + (threadIdx.x / 32) * offset_tile_x;
  int offset_y = blockIdx.y * P;
  int lane     = threadIdx.x % WARP_SIZE;

  int lanePlusOffsetX = lane + offset_x;

  /* Initialize v[] array */
  for(i=0; i<C; i++) {
    v[i] = dev_input[from2Dto1D(lanePlusOffsetX, i + offset_y, length)];
  }

  /* Main loop calculates for all P elements */
  #pragma unroll
  for(i=STRIDE; i<P+STRIDE; i++) {
    DATA_TYPE sum = 0;

    /* Left wing */
    for(j=-STRIDE; j<0; j++) {
      sum = v[i] * neighborCoefficient + sum;

      /* Shuffle up */
      sum = __shfl_up_sync(FULL_MASK, sum, 1);
    }

    /* Center column */
    for(j=-STRIDE; j<=STRIDE; j++) {
      if(j == 0)
        sum = v[i+j] * selfCoefficient + sum;
      else
        sum = v[i+j] * neighborCoefficient + sum;
    }

    /* Right wing */
    for(j=1; j<=STRIDE; j++) {
      /* Shuffle up */
      sum = __shfl_up_sync(FULL_MASK, sum, 1);
      sum = v[i] * neighborCoefficient + sum;
    }
    
    o[i] = sum;
  }

  /* Write the sum back to global memory */
  for(i=STRIDE; i<P+STRIDE; i++) {
    if(lane >= 2*STRIDE && lane+offset_x < length && i+offset_y < length-STRIDE) {
      dev_output[from2Dto1D(lane+offset_x-STRIDE, i+offset_y, length)] = o[i];
    }
  }
}

void read_input(DATA_TYPE **input, DATA_TYPE **output, string filename, int length)
{
  /* Define variables */
  int i, j;
  ifstream fp;

  /* Open input file */
  fp.open(filename.c_str(), ios::in);
  if(!fp) {
    cerr << "Couldn't open input file to read data!\n";
    exit(EXIT_FAILURE);
  }

  /* Allocate space for our arrays */
  *input = new DATA_TYPE[length * length];
  *output = new DATA_TYPE[length * length];

  /* Read data from file */
  for(i=0; i<length; i++) {
    for(j=0; j<length; j++) {
      fp >> (*input)[from2Dto1D(i, j, length)];
      (*output)[from2Dto1D(j, i, length)] = 0;
    }
  }
}

void write_output(DATA_TYPE *output, string filename, int length)
{
  /* Define variables */
  int i, j;
  ofstream fp;

  /* Open output file */
  fp.open(filename, ios::out);
  if(!fp) {
    cerr << "Couldn't open output file to write data!\n";
    exit(EXIT_FAILURE);
  }

  for(i=0; i<length; i++) {
    for(j=0; j<length; j++) {
      fp << output[from2Dto1D(j, i, length)] << " ";
    }
    fp << "\n";
  }
}

void copy_input_to_gpu(DATA_TYPE *input, DATA_TYPE **dev_input, DATA_TYPE **dev_output, int length)
{
  /* Allocate GPU memory for input and output arrays */
  gpuErrchk(hipMalloc((void**) dev_input, length * length * sizeof(DATA_TYPE)));
  gpuErrchk(hipMalloc((void**) dev_output, length * length * sizeof(DATA_TYPE)));

  /* Copy input array to GPU */
  gpuErrchk(hipMemcpy(*dev_input, input, length * length * sizeof(DATA_TYPE), hipMemcpyHostToDevice));
}