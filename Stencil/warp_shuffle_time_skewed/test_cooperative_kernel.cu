#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <iostream>

using namespace std;

int main() {
  int pi=0;
  hipDevice_t dev;
  hipDeviceGet(&dev,0); // get handle to device 0
  hipDeviceGetAttribute(&pi, hipDeviceAttributeCooperativeLaunch, dev);

  cout << "pi: " << pi << endl;

  return 0;
}
