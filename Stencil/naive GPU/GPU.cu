#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "timing.h"

using namespace std;

#define from2Dto1D(arr, x, y, length) ((arr)[(y)*length+(x)])
#define BLOCKX 32
#define BLOCKY 32

/* CPU Functions */
void stencil(int **dev_input, int **dev_output, int size, int stride, int length, int time);
__global__ void run_single_stencil(int *dev_input, int *dev_output, int true_size, int stride, int length);
__device__ int stencil_cross(int *arr, int x, int y, int length, int order);
void read_input(int **input, int **output, string filename, int length);
void write_output(int *output, string filename, int length);

/* GPU Functions */
void copy_input_to_gpu(int *input, int **dev_input, int **dev_output, int length);

int main(int argc, char *argv[])
{
  /* Define variables */
  int *input, *output;
  int *dev_input, *dev_output;
  int size, stride, length, time;
  string filename, output_filename;
  MyTimer timer;

  /* Check if the arguments are set */
  if(argc < 4) {
    cerr << "Usage: ./CPU <size> <stride> <time>\n";
    exit(EXIT_FAILURE);
  }

  /* Set initial variables */
  size             = atoi(argv[1]);
  stride           = atoi(argv[2]);
  time             = atoi(argv[3]);
  length           = 1 << size;       // length = 2 ^ size
  length          += 2 * stride;      //        + 2 * stride
  filename         = "../../Data/data_";
  filename        += to_string(size) + "_" + to_string(stride) + ".dat";
  output_filename  = "../../Data/gpu_";
  output_filename += to_string(size) + "_" + to_string(stride) + ".dat";

  /* Read data from input file */
  read_input(&input, &output, filename, length);

  /* Copy data to GPU */
  copy_input_to_gpu(input, &dev_input, &dev_output, length);

  /* Run Stencil */
  timer.StartTimer();
  stencil(&dev_input, &dev_output, size, stride, length, time);
  timer.StopTimer();

  /* Print duration */
  cout << "It took " << timer.GetDurationInSecondsAccurate() << " seconds to run!\n";

  /* Copy data back to CPU */
  hipMemcpy(dev_output, output, length * length * sizeof(int), hipMemcpyDeviceToHost);

  /* Output data */
  write_output(output, output_filename, length);

  /* Free allocated memory */
  hipFree(dev_input);
  hipFree(dev_output);
  delete [] input;
  delete [] output;

  /* End of program */
  return 0;
}

void stencil(int **dev_input, int **dev_output, int size, int stride, int length, int time)
{
  /* Define variables */
  int i;
  int **swap;
  int true_size = 1 << size;

  /* Loop over time dimension */
  for(i=0; i<time; i++) {
    /* Calculate block and grid sizes */
    dim3 block_size = dim3(BLOCKX, BLOCKY);
    dim3 grid_size = dim3((int)(length / BLOCKX) + 1, (int)(length / BLOCKY) + 1);
    run_single_stencil<<< grid_size, block_size >>>(*dev_input, *dev_output, true_size, stride, length);

    /* Swap pointers after each run so dev_output will always be output,
     * and dev_input will be always input
     */
    swap = dev_input;
    dev_input = dev_output;
    dev_output = swap;
  }
}

__global__ void run_single_stencil(int *dev_input, int *dev_output, int true_size, int stride, int length)
{
  /* Calculate indeces */
  int threadX = blockIdx.x * blockDim.x + threadIdx.x;
  int threadY = blockIdx.y * blockDim.y + threadIdx.y;

  /* Make sure indeces are not out of bound */
  if(threadX >= length || threadY >= length)
    return;

  /* Run single element stencil on all elements */
  from2Dto1D(dev_output, threadX, threadY, length) = stencil_cross(dev_input, threadX, threadX, length, stride);
}

__device__ int stencil_cross(int *arr, int x, int y, int length, int stride)
{
  /* Define variables */
  int sum = 0, i;

  /* Add cross pattern */
  for(i=-stride; i<=stride; i++) {
    sum += from2Dto1D(arr, x+i, y, length);
    sum += from2Dto1D(arr, x, y+i, length);
  }
  
  /* Counted center element twice, so substract it once */
  sum -= from2Dto1D(arr, x, y, length);

  /* Divide it by the number of elements */
  return sum / (stride * 4 + 1);
}

void read_input(int **input, int **output, string filename, int length)
{
  /* Define variables */
  int i, j;
  ifstream fp;

  /* Open input file */
  fp.open(filename.c_str(), ios::in);
  if(!fp) {
    cerr << "Couldn't open input file to read data!\n";
    exit(EXIT_FAILURE);
  }

  /* Allocate space for our arrays */
  *input = new int[length * length];
  *output = new int[length * length];

  /* Read data from file */
  for(i=0; i<length; i++) {
    for(j=0; j<length; j++) {
      fp >> from2Dto1D(*input, i, j, length);
    }
  }
}

void write_output(int *output, string filename, int length)
{
  /* Define variables */
  int i, j;
  ofstream fp;

  /* Open output file */
  fp.open(filename, ios::out);
  if(!fp) {
    cerr << "Couldn't open output file to write data!\n";
    exit(EXIT_FAILURE);
  }

  for(i=0; i<length; i++) {
    for(j=0; j<length; j++) {
      fp << from2Dto1D(output, i, j, length) << " ";
    }
    fp << "\n";
  }
}

void copy_input_to_gpu(int *input, int **dev_input, int **dev_output, int length)
{
  /* Allocate GPU memory for input and output arrays */
  hipMalloc((void**) dev_input, length * length * sizeof(int));
  hipMalloc((void**) dev_output, length * length * sizeof(int));

  /* Copy input array to GPU */
  hipMemcpy(input, *dev_input, length * length * sizeof(int), hipMemcpyHostToDevice);
}